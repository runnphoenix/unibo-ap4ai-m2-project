#include "hip/hip_runtime.h"
/****************************************************************************
*
 ****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <math.h>

#define R 3
const int BLKDIM = R;

__global__ void single_layer(float *x, int N, float *W, float *b, float *y)
{
  	int i = blockIdx.x;
    int j = threadIdx.x;

    if(i < N-R+1 && j < R)
    {
  		y[i] += x[i+j] * W[i * R + j];
  		printf("%.2f ", y[i]);
  		if(j == R-1)
        {
  			y[i] += *b;
  			printf("%.2f ", y[i]);
			y[i] = 1.0 / ( expf(-y[i]) + 1 );
			printf("%.2f \n", y[i]);
  		}
  		printf("%.2f \n", y[i]);
    }
}

int main( int argc, char *argv[] )
{
    int N = 10;
  	int K = 4;

    // get parameters from command line
    int c;
    while ((c = getopt (argc, argv, "n:k:")) != -1)
    {
        switch (c)
        {
            case 'n':
                N = atoi(optarg);
                break;
            case 'k':
                K = atoi(optarg);
                break;
        }
    }
    
    printf("%d %d\n", N, K);

    // Judge if the length of the k-th layer is bigger than 0
  	if (N - (K-1) * (R-1) <= 0) {
  		printf("The parameters you input couldn't support k layers. Please give bigger size of layer 0 or use less layers.\n");
  		return 0;
  	}

    // initialize the values of the first layer to 1
  	float x[N];
  	for (int i=0; i < N; i++) {
  		x[i] = -1.0;
  	}

    // create an activation
  	float activation[N];
    float *activation_d;
    hipMalloc((void**)&activation_d, N*sizeof(float));
  	memcpy(activation, x, N*sizeof(float));
    hipMemcpy(activation_d, activation, N*sizeof(float), hipMemcpyHostToDevice);

  	// start recording time
  	clock_t start = clock();

    // Loop over k layers
  	for(int t=1; t<K; t++) {
        // calculate length of this layer
        int layer_len = N - t * (R-1);

  		// initialize parameters b
  		//float b = rand() % 3 - 1;
        float b = 1.0;
        float *b_d;
        hipMalloc( (void**)&b_d, sizeof(float) );
        hipMemcpy(b_d, &b, sizeof(float), hipMemcpyHostToDevice);

        // parameter W
  		float W[layer_len][R];
        float *W_d;
        hipMalloc((void**)&W_d, layer_len*R*sizeof(float));
  		// random Initialization to range [-1,1]
  		#pragma omp parallel for collapse(2) num_threads(n_threads)
  		for (int i=0; i<layer_len; i++) {
  			for (int j=0; j<R; j++) {
  				//W[i][j] = ((rand() % 2000) - 1000) / 1000.0;
                  W[i][j] = 1.0 / 3;
  			}
  		}
        hipMemcpy(W_d, W, layer_len*R*sizeof(float), hipMemcpyHostToDevice);

        //y
        float y[layer_len];
        float *y_d;
        hipMalloc((void**)&y_d, layer_len*sizeof(float));
        for(int i=0; i<layer_len; i++){
            y[i] = 0.0;
        }
        hipMemcpy(y_d, y, layer_len*sizeof(float), hipMemcpyHostToDevice);

  		// do the calculation
  		single_layer<<<layer_len, BLKDIM>>>(activation_d, layer_len+R-1, W_d, b_d, y_d);
  		
        hipDeviceSynchronize();

        // copy result back
        hipMemcpy(y, y_d, layer_len*sizeof(float), hipMemcpyDeviceToHost);

        /*TEST
        printf("\nThe layer result got\n");
        for(int i=0; i<layer_len; i++){
            printf("%f ", y[i]);
        }
        printf("\n");
        */

  		// save the activation result
  		memcpy(activation, y, layer_len * sizeof(float));
        hipMemcpy(activation_d, activation, layer_len*sizeof(float), hipMemcpyHostToDevice);

        // free cuda memory
        hipFree(W_d); hipFree(y_d); hipFree(b_d);
  	}
  	
  	// calculate elapsed time
    clock_t end = clock();
    double time_elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Time elapsed: %.3f\n", time_elapsed);


 
	// print final result
	printf("\nFinal result is: ");
	int last_layer_len = N-(K-1)*(R-1);
	for(int i=0; i<last_layer_len; i++){
		printf("%.3f ", activation[i]);
	}
	printf("\n");

    return EXIT_SUCCESS;
}

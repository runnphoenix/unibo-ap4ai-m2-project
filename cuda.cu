/****************************************************************************
 * TODO
 * 1. extract layer_initialize function
 * 2. extract parameter_parsing function
 ****************************************************************************/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

#define R 3
const int BLKDIM = R;

__global__ void single_layer(float *x, int N, float *W, float *b, float *y)
{
  	int i = blockIdx.x;
    int j = threadIdx.x;

    __shared__ float local_y[R];

    if(i < N-R+1 && j < R)
    {
  		local_y[j] = x[i+j] * W[i * blockDim.x + j];
        printf("tidx %d %d x:%.2f W:%.2f y:%.2f \t", i, j, x[i+j], W[i * blockDim.x + j], local_y[j]);
    }

    __syncthreads();

    for(int k=0; k<R; k++)
    {
        y[i] += local_y[k];
    }

    if(j == R-1)
    {
        y[i] += *b;
        y[i] = 1.0 / (exp(-y[i]) + 1);
    }
}

int main( int argc, char *argv[] )
{
    int N = 10;
  	int K = 4;

    // get parameters from command line
    int c;
    while ((c = getopt (argc, argv, "n:k:")) != -1)
    {
        switch (c)
        {
            case 'n':
                N = atoi(optarg);
                break;
            case 'k':
                K = atoi(optarg);
                break;
        }
    }

    printf("%d %d\n", N, K);

    // Judge if the length of the k-th layer is bigger than 0
  	if (N - (K-1) * (R-1) <= 0) {
  		printf("The parameters you input couldn't support k layers. Please give bigger size of layer 0 or use less layers.\n");
  		return 0;
  	}

    // initialize the values of the first layer to 1
  	float x[N];
  	for (int i=0; i < N; i++) {
  		x[i] = -1.0;
  	}

    // create an activation
  	float activation[N];

    float *activation_d;
    hipMalloc((void**)&activation_d, N*sizeof(float));
  	memcpy(activation, x, N*sizeof(float));
    hipMemcpy(activation_d, activation, N*sizeof(float), hipMemcpyHostToDevice);

  	// start recording time

    // Loop over k layers
  	for(int t=1; t<K; t++) {
        // calculate length of this layer
        int layer_len = N - t * (R-1);

  		// initialize parameters b
  		//float b = rand() % 3 - 1;
        float b = 1.0;
        float W[layer_len][R];
        float y[layer_len];


        float *b_d;
        float *W_d;
        float *y_d;


        hipMalloc((void**)&b_d, sizeof(float));
        hipMalloc((void**)&W_d, layer_len*R*sizeof(float));
        hipMalloc((void**)&y_d, layer_len*sizeof(float));


  		// random Initialization to range [-1,1]
  		#pragma omp parallel for collapse(2) num_threads(n_threads)
  		for (int i=0; i<layer_len; i++) {
  			for (int j=0; j<R; j++) {
  				//W[i][j] = ((rand() % 2000) - 1000) / 1000.0;
                  W[i][j] = 1.0 / 3;
  			}
  		}
        for(int i=0; i<layer_len; i++){
            y[i] = 0.0;
        }


        hipMemcpy(b_d, &b, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(W_d, W, layer_len*R*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(y_d, y, layer_len*sizeof(float), hipMemcpyHostToDevice);

  		// do the calculation
  		single_layer<<<layer_len, BLKDIM>>>(activation_d, layer_len+R-1, W_d, b_d, y_d);

        hipDeviceSynchronize();

        // copy result back
        hipMemcpy(y, y_d, layer_len*sizeof(float), hipMemcpyDeviceToHost);

  		// save the activation result
  		memcpy(activation, y, layer_len * sizeof(float));
        hipMemcpy(activation_d, activation, layer_len*sizeof(float), hipMemcpyHostToDevice);

        // free cuda memory
        hipFree(W_d); hipFree(y_d); hipFree(b_d);
  	}

	// print final result
	printf("\nFinal result is: ");
	int last_layer_len = N-(K-1)*(R-1);
	for(int i=0; i<last_layer_len; i++){
		printf("%.3f ", activation[i]);
	}
	printf("\n");

    return EXIT_SUCCESS;
}

/****************************************************************************
 * cuda_bigger_block.cu - a simple multi-layer Nerual Network
 *
 * Assignment of Module 2 of Ap4AI course of AI master degree @unibo
 *
 * Last updated in 2021 by Hanying Zhang <hanying.zhang@studio.unibo.it>
 * 
 * To the extent possible under law, the author(s) nave dedicated all
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * --------------------------------------------------------------------------
 *
 * compile with:
 * nvcc cuda_bigger_block.c -o cuda_bigger_block
 *
 * Run with:
 * ./cuda_bigger_block -n # of nodes -k # of layers
 * 
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define R 3
const int BLKDIM = (64/R*R)*R;

/*  BLKDIM optimization
 *  # of threads shoule be able to divide (32 * R)
 *  (n_node / R * R) is # of nodes being able to divide R  ->
 *  (n_node / R * R) * R is # of threads being able to divide R
 */

// Use a __device__ function to calculate Sigmoid
__device__ float Sigmoid(float x, float b)
{
    return 1.0 / ( expf(-x - b) + 1 );
}

/* The calculation of y values for one layer */
__global__ void one_layer_calc(float *x, float *W, float *b, float *y, int N)
{
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;  // global thread index
    int lidx = threadIdx.x;                            // local thread index
    int gi = gidx / R;            // global node index
    int li = lidx / R;            // local node index
    int j = gidx - gi * R;        // index of related values in previous layer for each value in y

    int layer_len = N - R + 1; 
 
    float y_tmp = 0.0;
    // shared memory used to store local values in y
    __shared__ float local_y[BLKDIM];

    if(gi < layer_len && j < R) {
        local_y[lidx] = x[gi + j] * W[gi * R + j];
        //printf("i:%d j:%d lidx: %d x:%.2f W:%.2f y:%.2f \n", \
                  gi, j, lidx, x[gi+j], W[gi * R + j], local_y[lidx]);
    }

    __syncthreads();
    //printf("\n");  
    
    // Accumulate R values of each node in y
    if(gi < layer_len && j < R){
    	for (int p=0; p<R; p++) {
        	y_tmp += local_y[li * R + p];
        	//printf("i:%d j:%d lidx: %d local_y:%.2f tmp:%.2f \n", gi,j,lidx, local_y[li * R + p], y_tmp);
    	}
    }

    //__syncthreads();
   
    if(gi < layer_len){
    	// Sigmoid
    	y_tmp = Sigmoid(y_tmp, *b);
    	// Copy temp values to y
    	*(y+gi) = y_tmp;
    }
}

/* Random values between -1 and 1 */
float random_init_small()
{
	return ((rand() % 2000) - 1000) / 1000.0;     // random Initialization to values in range [-1,1]
}

/* Initialize the W and b parameters for one layer */
//TODO: change to initialize all parameters

/* Read in the network parameters (N, K) from command-line input. */
void parse_command_line_parameters(int argc, char *argv[], int *N, int *K)
{
    int c;
    while ((c = getopt (argc, argv, "n:k:")) != -1) {
        switch (c) {
            case 'n': // N
                *N = atoi(optarg);
                break;
            case 'k': // K
                *K = atoi(optarg);
                break;
        }
    }
}

int main( int argc, char *argv[] )
{
	srand(42);
	
    int N = 100;
    int K = 3;

    // get N, K from command line
    parse_command_line_parameters(argc, argv, &N, &K);
    printf("input size:%d, number of layers:%d.\n",  N, K);

    // Judge if the length of the k-th layer is bigger than 0
    int last_layer_len = N - (K-1) * (R-1);
    if (last_layer_len <= 0) {
        printf("The parameters you input couldn't support k layers. \
                Please give bigger size of layer 0 or use less layers.\n");
        return EXIT_FAILURE;
    } 

	// create an array which stores all the layer-values of w, b and y
	int first_layer_len = N; // input included
	int total_b_len = K - 1;
	int total_y_len = K * (first_layer_len + last_layer_len) / 2;
	int total_W_len = (total_y_len - N) * R;
	
	float *b = (float*) malloc(total_b_len * sizeof(float));
	float *y = (float*) malloc(total_y_len * sizeof(float));
	float *W = (float*) malloc(total_W_len * sizeof(float));
	
	// initialize the values of y, w and b
	for (int i=0; i < total_y_len; i++) {
        y[i] = random_init_small();
    }
    for (int i=0; i < K-1; i++) {
		b[i] = random_init_small();
    }
    for (int i=0; i < total_W_len; i++) {
		W[i] = random_init_small();
    }
    
    /*/TEST
    for (int i=0; i < total_y_len; i++) {
        printf("%.2f ", y[i]);
    }
    printf("\n");
    for (int i=0; i < K-1; i++) {
		printf("%.2f ", b[i]);
    }
    printf("\n");
    for (int i=0; i < total_W_len; i++) {
		printf("%.2f ", W[i]);
    }
    printf("\n");
    */
    
	
	// create gpu related b, w and y
	float *b_d;
    float *W_d;
    float *y_d;
    
    // Start recording time costage
    clock_t start = clock();
	
	// cuda memcpy
	hipMalloc( (void**)&b_d, (K-1)       * sizeof(float) );
    hipMalloc( (void**)&W_d, total_W_len * sizeof(float) );
    hipMalloc( (void**)&y_d, total_y_len * sizeof(float) );

    hipMemcpy(b_d, b, (K-1)       * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(W_d, W, total_W_len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, total_y_len * sizeof(float), hipMemcpyHostToDevice);

    // Loop over K layers
    for(int k=1; k<K; k++) {
        // calculate lengthes of this layer and the previous layer
        int layer_len = N - k * (R-1);
        int in_layer_len = layer_len + R - 1;

        // calculation of each layer
        // printf("\nGRIDDIM %d BLKDIM: %d\n", (layer_len*R+BLKDIM-1)/BLKDIM, BLKDIM);
        int y_start_idx = k * (N + N - (k-1)*(R-1)) / 2;
        int x_start_idx = (k-1) * (N + N - (k-2)*(R-1)) / 2;
        int W_start_idx = (y_start_idx-N) * R;
        
        one_layer_calc<<<(layer_len*R+BLKDIM-1)/BLKDIM, BLKDIM>>>(y_d + x_start_idx, W_d + W_start_idx, \
                                                                  b_d + (k-1), y_d + y_start_idx, in_layer_len);

        hipDeviceSynchronize();
    }
    
    // copy result back to host
    hipMemcpy(y, y_d, total_y_len * sizeof(float), hipMemcpyDeviceToHost);

    // print final result
    printf("\nFinal result is: ");
    for(int i=(total_y_len - last_layer_len); i<total_y_len; i++) {
        printf("%.3f ", y[i]);
    }
    printf("\n");
    
    // calculate elapsed time
    clock_t end = clock();
    double time_elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Elapsed time: %.3fs\n", time_elapsed);
    
    
    // Free memory
    hipFree(W_d); hipFree(y_d); hipFree(b_d);  // free cuda memory
    free(b); free(W); free(y);                    // free heap memory

    return EXIT_SUCCESS;
}

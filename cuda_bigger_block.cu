/****************************************************************************
 * cuda_bigger_block.cu - a simple multi-layer Nerual Network
 *
 * Assignment of Module 2 of Ap4AI course of AI master degree @unibo
 *
 * Last updated in 2021 by Hanying Zhang <hanying.zhang@studio.unibo.it>
 * 
 * To the extent possible under law, the author(s) nave dedicated all
 * copyright and related and neighboring rights to this software to the 
 * public domain worldwide. This software is distributed without any warranty.
 *
 * --------------------------------------------------------------------------
 *
 * compile with:
 * nvcc cuda_bigger_block.c -o cuda_bigger_block
 *
 * Run with:
 * ./cuda_bigger_block -n # of nodes -k # of layers
 * 
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define R 3
const int BLKDIM = (64/R*R)*R;

/*  BLKDIM optimization
 *  # of threads shoule be able to divide (32 * R)
 *  (n_node / R * R) is # of nodes being able to divide R  ->
 *  (n_node / R * R) * R is # of threads being able to divide R
 */

// Use a __device__ function to calculate Sigmoid
__device__ float Sigmoid(float x, float b)
{
    return 1.0 / ( expf(-x - b) + 1 );
}

/* The calculation of y values for one layer */
__global__ void one_layer_calc(float *x, float *W, float *b, float *y, int N)
{
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;  // global thread index
    int lidx = threadIdx.x;                            // local thread index
    int gi = gidx / R;            // global node index
    int li = lidx / R;            // local node index
    int j = gidx - gi * R;        // index of related values in previous layer for each value in y

    int pre_layer_len = N - R + 1; 
 
    float y_tmp = 0.0;
    // shared memory used to store local values in y
    __shared__ float local_y[BLKDIM];

    if(gi < pre_layer_len && j < R) {
        local_y[lidx] = x[gi + j] * W[gi * R + j];
        //printf("i:%d j:%d lidx: %d x:%.2f W:%.2f y:%.2f \n", \
                  gi, j, lidx, x[gi+j], W[gi * R + j], local_y[lidx]);
    }

    __syncthreads();
    //printf("\n");  
    
    // Accumulate R values of each node in y
    for (int p=0; p<R; p++) {
        y_tmp += local_y[li * R + p];
        //printf("i:%d j:%d lidx: %d local_y:%.2f tmp:%.2f \n", gi,j,lidx, local_y[li * R + p], y_tmp);
    }

    __syncthreads();
   
    // Sigmoid
    y_tmp = Sigmoid(y_tmp, *b);

    // Copy temp values to y
    y[gi] = y_tmp;
}

/* Initialize the W and b parameters for one layer */
//TODO: change to initialize all parameters
void init_layer_parameters(float (*W)[R], float w_v, float *b, float b_v, int layer_len)
{
    for (int i=0; i<layer_len; i++) {
        for (int j=0; j<R; j++) {
            W[i][j] = w_v;
        }
    }

    *b = b_v;
}

/* Read in the network parameters (N, K) from command-line input. */
void parse_command_line_parameters(int argc, char *argv[], int *N, int *K)
{
    int c;
    while ((c = getopt (argc, argv, "n:k:")) != -1) {
        switch (c) {
            case 'n': // N
                *N = atoi(optarg);
                break;
            case 'k': // K
                *K = atoi(optarg);
                break;
        }
    }
}

int main( int argc, char *argv[] )
{
    int N = 100;
    int K = 3;

    // get N, K from command line
    parse_command_line_parameters(argc, argv, &N, &K);
    printf("input size:%d, number of layers:%d.\n",  N, K);

    // Judge if the length of the k-th layer is bigger than 0
    int last_layer_len = N - (K-1) * (R-1);
    if (last_layer_len <= 0) {
        printf("The parameters you input couldn't support k layers. \
                Please give bigger size of layer 0 or use less layers.\n");
        return EXIT_FAILURE;
    } 

	// create an array which stores all the layer-values of w, b and y
	int first_layer_len = N; // input included
	int total_b_len = K - 1;
	int total_y_len = K * (first_layer_len + last_layer_len) / 2;
	int total_W_len = total_y_len * R;
	
	float *b = (float*)malloc(total_b_len * sizeof(float));
	float *y = (float*)malloc(total_y_len * sizeof(float));
	float *W = (float*)malloc(total_W_len * sizeof(float));
	
	// initialize the values of y, w and b
	//float b_v = rand() % 3 - 1;
    //float W_v = ((rand() % 2000) - 1000) / 1000.0;
    //init_layer_parameters(W, W_v, &b, b_v, layer_len);
	for (int i=0; i < total_y_len; i++) {
		if(i < N) {
        	y[i] = -1.0;
        }
        else {
        	y[i] = 0.0;
        }
    }
    for (int i=0; i < K-1; i++) {
		b[i] = 1.0;
    }
    for (int i=0; i < total_y_len * R; i++) {
		W[i] = 1.0 / 3;
    }
	
	// create gpu related b, w and y
	float *b_d;
    float *W_d;
    float *y_d;
    
    // Start recording time costage
    clock_t start = clock();
	
	// cuda memcpy
	hipMalloc( (void**)&b_d, (K-1)       * sizeof(float) );
    hipMalloc( (void**)&W_d, total_W_len * sizeof(float) );
    hipMalloc( (void**)&y_d, total_y_len * sizeof(float) );

    hipMemcpy(b_d, b, (K-1)       * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(W_d, W, total_W_len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, total_y_len * sizeof(float), hipMemcpyHostToDevice);

    // Loop over K layers
    for(int k=1; k<K; k++) {
        // calculate lengthes of this layer and the previous layer
        int layer_len = N - k * (R-1);
        int in_layer_len = layer_len + R - 1;

        // calculation of each layer
        // printf("\nGRIDDIM %d BLKDIM: %d\n", (layer_len*R+BLKDIM-1)/BLKDIM, BLKDIM);
        int y_start_idx = k * (N + N - (k-1)*(R-1)) / 2;
        int x_start_idx = (k-1) * (N+N-(k-2)*(R-1)) / 2;
        int W_start_idx = y_start_idx * R;
        
        one_layer_calc<<<(layer_len*R+BLKDIM-1)/BLKDIM, BLKDIM>>>(y_d + x_start_idx, W_d + W_start_idx, b_d + (k-1), y_d + y_start_idx, in_layer_len);

        hipDeviceSynchronize();
    }
    
    // copy result back to host
    hipMemcpy(y, y_d, total_y_len * sizeof(float), hipMemcpyDeviceToHost);
    // Free cuda memory
    hipFree(W_d); hipFree(y_d); hipFree(b_d);

    // calculate elapsed time
    clock_t end = clock();
    double time_elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Time elapsed: %.3f\n", time_elapsed);

    hipFree(b_d); hipFree(W_d); hipFree(y_d);

    // print final result
    printf("\nFinal result is: ");
    for(int i=(total_y_len - last_layer_len); i<total_y_len; i++) {
        printf("%.3f ", y[i]);
    }
    printf("\n");

    return EXIT_SUCCESS;
}
